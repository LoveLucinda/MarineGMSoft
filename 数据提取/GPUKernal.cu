#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include "stdafx.h"
#include "time.h"
#include "math.h"
/**
  * CUDA����˺���
  *
  */
/*===========================================================================================================
 �������ܣ�
 ������

����˵����GPU����˺�����cpu�޷�����
		  ÿһ���̺߳Ŷ�Ӧ�ڼ������ÿһ�����������˻�
		  #pragma unroll�����forѭ���Ǵ��еģ������ۼ���ͣ����д��id��Ӧ������Ԫ��
============================================================================================================*/
__global__ void
	DemCorrect(const double *demdata,const GRDRANGEINDEX* indexarray,double *CorrectData,int numberofelement,GRDSTRUCT datainfo)
{
	int id=blockDim.x*blockIdx.x+threadIdx.x;		//�̺߳ţ��൱��CPUѭ�����i��
	if(id>=numberofelement)return;

	double temp=0;
	double grav=0,density=1.64;
	double tempx1,tempx2,tempy1,tempy2,G122,G121,G112,G111,G222,G221,G212,G211;
	double p222,p221,p212,p211,p122,p121,p112,p111;		//��x��y��z��˳������
	double p2,p1;
	double z=-0.1;
	double x1=0,x2=0,y1=0,y2=0,z1=0,z2=0;
#pragma unroll							//��һ���̼߳���һ�����ĸ���ֵ�����ݣ�
	for (int i = indexarray[id].rowmin; i <indexarray[id].rowmax; i++)
	{
		for (int j = indexarray[id].colmin; j < indexarray[id].colmax; j++)
		{
			

		//ģ�Ͳ���
		/*x1=datainfo.xmin+j*datainfo.dx,x2=datainfo.xmin+(j+1)*datainfo.dx,y1=datainfo.ymin+i*datainfo.dy,y2=datainfo.ymin+(i+1)*datainfo.dy,
			z1=0-z,z2=demdata[j+i*datainfo.cols]-z;*/
		
		//�м����
		
		p2=z2*z2;	//X�������ʽ���ǳ���
		p1=z1*z1;

		tempx1=x1-indexarray[id].x0;		//���������
		tempx2=x2-indexarray[id].x0;
		
		tempy1=y1-indexarray[id].y0;
		tempy2=y2-indexarray[id].y0;
			
		p222=sqrt(pow(tempx2,2)+pow(tempy2,2)+p2);		//����
		p221=sqrt(pow(tempx2,2)+pow(tempy2,2)+p1);
		p212=sqrt(pow(tempx2,2)+pow(tempy1,2)+p2);
		p211=sqrt(pow(tempx2,2)+pow(tempy1,2)+p1);

		////
		p122=sqrt(pow(tempx1,2)+pow(tempy2,2)+p2);		//����
		p121=sqrt(pow(tempx1,2)+pow(tempy2,2)+p1);
		p112=sqrt(pow(tempx1,2)+pow(tempy1,2)+p2);
		p111=sqrt(pow(tempx1,2)+pow(tempy1,2)+p1);
		////	
		G222=tempx2*log(tempy2+p222)+tempy2*log(tempx2+p222)+z2*atan2((z2*p222),(tempx2*tempy2));		//������
		G221=tempx2*log(tempy2+p221)+tempy2*log(tempx2+p221)+z1*atan2((z1*p221),(tempx2*tempy2));
		G212=tempx2*log(tempy1+p212)+tempy1*log(tempx2+p212)+z2*atan2((z2*p212),(tempx2*tempy1));
		G211=tempx2*log(tempy1+p211)+tempy1*log(tempx2+p211)+z1*atan2((z1*p211),(tempx2*tempy1));
		
		G122=tempx1*log(tempy2+p122)+tempy2*log(tempx1+p122)+z2*atan2((z2*p122),(tempx1*tempy2));
		G121=tempx1*log(tempy2+p121)+tempy2*log(tempx1+p121)+z1*atan2((z1*p121),(tempx1*tempy2));
		G112=tempx1*log(tempy1+p112)+tempy1*log(tempx1+p112)+z2*atan2((z2*p112),(tempx1*tempy1));
		G111=tempx1*log(tempy1+p111)+tempy1*log(tempx1+p111)+z1*atan2((z1*p111),(tempx1*tempy1));
			
		grav+=-6.67*0.001*density*(G222+G211+G121+G112-G221-G212-G122-G111);	//mGal

		}
		
	}
	CorrectData[id]=grav;
}

/*===========================================================================================================
 �������ܣ��������ϵĵ��θ���ֵ��������������У����
 ������
	
����˵����GPU����˺�����cpu�޷�����
		  ÿһ���̺߳Ŷ�Ӧ�ڼ������ÿһ�����������˻�
		  #pragma unroll�����forѭ���Ǵ��еģ������ۼ���ͣ����д��id��Ӧ������Ԫ��
============================================================================================================*/
extern "C"
void GPU_Correct(double* CorrectData,int CorrectDataNumber,double* DemData,GRDRANGEINDEX* IndexArray,GRDSTRUCT demdatainfo)
{
	//CUDA���������
	hipError_t err=hipSuccess;
	//����GPU��������
	size_t size=CorrectDataNumber*sizeof(double);
	double *d_CorrectData=NULL;
	err=hipMalloc((void**)&d_CorrectData,size);
	if (err!=hipSuccess){AfxMessageBox(_T("����GPU����d_CorrectDataʧ��\n"));exit(EXIT_FAILURE);}

	size_t size_indexarray=CorrectDataNumber*sizeof(GRDRANGEINDEX);
	GRDRANGEINDEX *d_IndexArray=NULL;
	err=hipMalloc((void**)&d_IndexArray,size_indexarray);
	if (err!=hipSuccess){AfxMessageBox(_T("����GPU����d_IndexArrayʧ��\n"));exit(EXIT_FAILURE);}

	size_t size_demdata=demdatainfo.rows*demdatainfo.cols*sizeof(double);
	double *d_DemData=NULL;
	err=hipMalloc((void**)&d_DemData,size_demdata);
	if (err!=hipSuccess){AfxMessageBox(_T("����GPU����d_DemDataʧ��\n"));exit(EXIT_FAILURE);}

	//��host�洢�ռ俽����GPU
	//AfxMessageBox(_T("host�洢�ռ俽����GPU\n"));
	err=hipMemcpy(d_CorrectData,CorrectData,size,hipMemcpyHostToDevice);
	if (err!=hipSuccess){AfxMessageBox(_T("�����洢�ռ�CorrectData��deviceʧ��\n"));exit(EXIT_FAILURE);}

	err=hipMemcpy(d_IndexArray,IndexArray,size_indexarray,hipMemcpyHostToDevice);
	if (err!=hipSuccess){AfxMessageBox(_T("�����洢�ռ�IndexArray��deviceʧ��\n"));exit(EXIT_FAILURE);}

	err=hipMemcpy(d_DemData,DemData,size_demdata,hipMemcpyHostToDevice);
	if (err!=hipSuccess){AfxMessageBox(_T("�����洢�ռ�DemData��deviceʧ��\n"));exit(EXIT_FAILURE);}


	//����CUDA�˺���
	int threadsPerBlock=512;//��ȡ�豸����
	int blockPerGrid=(CorrectDataNumber+threadsPerBlock-1)/threadsPerBlock;
	//printf("CUDA�˺������� %d ��blocks��%d ���߳�\n",blockPerGrid,threadsPerBlock);
	DemCorrect<<<blockPerGrid,threadsPerBlock>>>(d_DemData,d_IndexArray,d_CorrectData,CorrectDataNumber,demdatainfo);		//GPU����˺���
	err=hipGetLastError();
	if (err!=hipSuccess){AfxMessageBox(_T("����CUDA�˺���ʧ�ܣ�������룺%s)��\n"));exit(EXIT_FAILURE);}

	
	//��GPU����������ݵ��ڴ�
	//printf("��GPU����������ݵ��ڴ�\n");
	err=hipMemcpy(CorrectData,d_CorrectData,size,hipMemcpyDeviceToHost);
	CString errorstr;errorstr.Format(_T("��GPU��������ʧ�ܣ�������룺%s)��\n"),hipGetErrorString(err));
	if (err!=hipSuccess){AfxMessageBox(errorstr);exit(EXIT_FAILURE);}

	//�ͷ��ڴ�
	err=hipFree(d_CorrectData);
	if (err!=hipSuccess){AfxMessageBox(_T("CUDA�ͷ�d_A�洢�ռ�ʧ�ܣ�������룺%s)��\n"));exit(EXIT_FAILURE);}
	err=hipFree(d_IndexArray);
	if (err!=hipSuccess){AfxMessageBox(_T("CUDA�ͷ�d_B�洢�ռ�ʧ�ܣ�������룺%s)��\n"));exit(EXIT_FAILURE);}
	err=hipFree(d_DemData);
	if (err!=hipSuccess){AfxMessageBox(_T("CUDA�ͷ�d_C�洢�ռ�ʧ�ܣ�������룺%s)��\n"));exit(EXIT_FAILURE);}

	//����device
	err=hipDeviceReset();
	if (err!=hipSuccess){AfxMessageBox(_T("����deviecʧ�ܣ�������룺%s)��\n",));exit(EXIT_FAILURE);}
}
/*===========================================================================================================
 �������ܣ���ȡ����ӡ�豸����
============================================================================================================*/
extern "C"
void GetGPUProperty()
{
	//1.1 ��ȡGPU����
	int deviceCount;
	hipGetDeviceCount(&deviceCount);
	
	//1.2 ��ȡGPU�豸����
	int devID=0;			//�豸��
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp,devID);//deviceProp.major ��deviceProp.minor �ֱ�ΪCUDA�豸�������������汾�źʹΰ汾�š�CPU��������ֵ������Ϊ9999

	//--1.2.1 �豸��ȫ�ִ洢����С
	int TotalGlobalMem=deviceProp.totalGlobalMem;//�豸�е�ȫ�ִ洢����С��global memory��������Ƕ����Կ������ǰ����Դ�Ĵ�С��
	//--1.2.2 SM����
	int SMcount=deviceProp.multiProcessorCount;//�豸�п��õ�����˴�����SM�ĸ�����ÿ��SM��8��SP,����SP�ĸ���Ϊ��8* deviceProp.multiProcessorCout
	int SPcount=SMcount*8;
	//--1.2.3 �����洢����С
	int ConstMem=deviceProp.totalConstMem;//�����洢���Ĵ�С
	//--1.2.4
	int ShareMem=deviceProp.sharedMemPerBlock;//ÿ��block�п��õĹ���洢����С
	//--1.2.5
	int RegsCount=deviceProp.regsPerBlock;//ÿ��block�п��õļĴ�������
	//--1.2.6 
	int ThreadCount=deviceProp.warpSize;//�߳���warp�ɶ��ٸ�thread���
	//--1.2.7
	int ThreadInBlock=deviceProp.maxThreadsPerBlock;//һ��block���������ж��ٸ�thread

	//1.3 ���GPU��Ϣ
	CString gpupropstr,tempstr;
	gpupropstr.Format(_T("����GPU��������: %d ��\n"),deviceCount);
	tempstr.Format(_T("ȫ�ִ洢����С(Mb):%d\nThread������%d\n"),TotalGlobalMem/1024/1024,ThreadCount);
	gpupropstr+=tempstr;
	tempstr.Format(_T("ÿ��block��Thread������%d\nÿ��block�п��üĴ���������%d\nSP������%d\n"),ThreadInBlock,RegsCount,SPcount);
	gpupropstr+=tempstr;
	AfxMessageBox(gpupropstr);
	printf("ÿ��block�п��ù���洢����С��%d\n",ShareMem);
	printf("����������С��%d\n",ConstMem);
	printf("SM������%d\n",SMcount);
}
